// defines
#define REAL double
#define BLOCK_SIZE_X 2
#define BLOCK_SIZE_Y 2
#define BLOCK_SIZE_Z 16
#define TIMECODE 1

// includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

#include "services.cu"
#include "structures.cu"
#include "inout.cu"
#include "diffusion.cu"
#include "precipitates.cu"


//////////////////
// Program main //
//////////////////

// Host: CPU  --> variables with _h
// Device: GPU --> variables with _d


int main(int argc, char **argv)
{

    //----------------------------------------------------//
    // choose the device (GPU) and display its properties //
    //----------------------------------------------------//
    hipSetDevice(atoi(argv[2]));
    DisplayDeviceProperties(atoi(argv[2]));


    //---------------------//
    // Read parameter file //
    //---------------------//
    Parameters par_h[1];
    ReadInputFile(argv[1], par_h);
    DisplayParams(par_h);

    Parameters *par_d;    
    hipMalloc((void**)&par_d, sizeof(Parameters));
    hipMemcpy(par_d, par_h, sizeof(Parameters), hipMemcpyHostToDevice);

    //------------//
    // Memory CPU //
    //------------//
    size_t RealSize = par_h[0].Nx * par_h[0].Ny * par_h[0].Nz * sizeof(REAL);
    REAL *U_h =(REAL *) malloc(RealSize);
    REAL *Phi_h =(REAL *) malloc(RealSize);
    Precipitate *ppts_h = (Precipitate *) malloc(sizeof(Precipitate)*par_h[0].Nppts);
    
    //------------//
    // Memory GPU //
    //------------//
    REAL *U_d, *Utemp_d, *Ubuff_d;
    REAL *Phi_d;
    hipMalloc((void**)&U_d, RealSize);
    hipMalloc((void**)&Utemp_d, RealSize);
    hipMalloc((void**)&Phi_d, RealSize);

    Precipitate *ppts_d;
    hipMalloc((void**)&ppts_d,sizeof(Precipitate)*par_h[0].Nppts);

    // GPU blocks for diffusion
    dim3 DimBlocks(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);
    dim3 NumBlocks(par_h[0].Nx/BLOCK_SIZE_X, par_h[0].Ny/BLOCK_SIZE_Y, par_h[0].Nz/BLOCK_SIZE_Z);
    
    // GPU blocks for ppts
    dim3 DimBlocksPpts(1);
    dim3 NumBlocksPpts(par_h[0].Nppts);


    //---------------------------//
    // set initial configuration //
    //---------------------------//
    
    srand(par_h[0].seed); // initialize random seed
    
    InitializeField<<<NumBlocks,DimBlocks>>>(U_d, par_d);
    hipMemcpy(U_h, U_d, RealSize, hipMemcpyDeviceToHost);

    InitializePrecipitates(ppts_h, par_h);
    hipMemcpy(ppts_d, ppts_h, sizeof(Precipitate)*par_h[0].Nppts, hipMemcpyHostToDevice);

    UpdateFields<<<NumBlocksPpts,DimBlocksPpts>>>(U_d, Phi_d, ppts_d, par_d);
    hipMemcpy(Phi_h, Phi_d, RealSize, hipMemcpyDeviceToHost);
    hipMemcpy(U_h, U_d, RealSize, hipMemcpyDeviceToHost);
    hipMemcpy(ppts_h, ppts_d, sizeof(Precipitate)*par_h[0].Nppts, hipMemcpyDeviceToHost);


    //----------------------//
    // Output initial state //
    //----------------------//
    int index=0;
    int iter=0;
    WriteUVtk(index, U_h, par_h, iter);
    WritePhiVtk(index, Phi_h, par_h, iter);
    WritePpts(index, ppts_h, par_h, iter);
    WriteProps(par_h, U_h, Phi_h, ppts_h, iter, 1);
    
    
    //-----------------//
    // timer variables //
    //-----------------//
#if(TIMECODE)
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float timer;
    hipEventRecord(start, 0);
#endif


    for(iter=1; iter<=par_h[0].Nsteps; iter++) 
    {    

        //-----------//
        // Diffusion //
        //-----------//  
        Diffusion<<<NumBlocks,DimBlocks>>>(U_d, Utemp_d, Phi_d, par_d);
        Ubuff_d=U_d;     U_d=Utemp_d;      Utemp_d=Ubuff_d;
        
        //------------//
        // PPT growth //
        //------------//  
        PrecipitatesGrowth<<<NumBlocksPpts,DimBlocksPpts>>>(U_d, Phi_d, ppts_d, par_d);
        
        //-------------//
        // PPT overlap //
        //-------------// 
        PrecipitatesOverlap<<<NumBlocksPpts,DimBlocksPpts>>>(U_d, Phi_d, ppts_d, par_d);
        
        //------------//
        // PPT upadate //
        //------------//        
        UpdateFields<<<NumBlocksPpts,DimBlocksPpts>>>(U_d, Phi_d, ppts_d, par_d);
        
        
        //--------//
        // Output //
        //--------//  
        
        if((iter%par_h[0].every)==0)
        {
            hipMemcpy(U_h, U_d, RealSize, hipMemcpyDeviceToHost);
            hipMemcpy(Phi_h, Phi_d, RealSize, hipMemcpyDeviceToHost);
            hipMemcpy(ppts_h, ppts_d, sizeof(Precipitate)*par_h[0].Nppts, hipMemcpyDeviceToHost);

            WriteProps(par_h, U_h, Phi_h, ppts_h, iter, 0);            
            
            if((iter%(par_h[0].Nsteps/par_h[0].NOutput))==0)
            {
                index=index+1;
                WriteUVtk(index, U_h, par_h, iter);
                WritePhiVtk(index, Phi_h, par_h, iter);
                WritePpts(index, ppts_h, par_h, iter);
            } 
        }
        

    }
    
    
    hipError_t err = hipGetLastError();  // add
    if (err != hipSuccess) std::cout << "CUDA error: " << hipGetErrorString(err) << std::endl; // add
    hipProfilerStop();
    
    #if(TIMECODE)
        hipEventRecord(stop, 0); 
        hipEventSynchronize(stop); 
        hipEventElapsedTime(&timer, start, stop);
        timer=timer/1000.0;
        FILE *OutFile=fopen("timer.out","w");    
        fprintf(OutFile, "#total_time_sec\t #time_perGP_perTS_sec\n");
        fprintf(OutFile, "%.8g \t %.8g\n",timer,(timer/(REAL(par_h[0].Nx*par_h[0].Ny))/(REAL(par_h[0].Nsteps))));
        fclose(OutFile);
        printf("time=%f sec\n",timer);
    #endif
                                    
    return 0;

}

